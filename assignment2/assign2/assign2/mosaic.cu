#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

struct EveryPixel {
	unsigned char r, g, b;
} ;

struct WholeImage {
	int x, y;
	EveryPixel *d;
} ;

#define FAILURE 0
#define SUCCESS !FAILURE

#define USER_NAME "acp18gc"		//replace with your user name

void print_help();
char *filename, *modename, *outname, *option;

int process_command_line(int argc, char *argv[]);

typedef enum MODE { CPU, OPENMP, CUDA, ALL } MODE;

unsigned int c;
MODE execution_mode = CUDA;

WholeImage *image;
char buffer[16];

int readFile();

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//writeFile function
void writeFile(const char *filename, WholeImage *img, int target, const char *option) {
	int tp = 6;

	if (option == NULL)
	{
		tp = 6;
	}
	else if (!strcmp(option, "PPM_BINARY")) {
		tp = 6;
	}
	else {
		tp = 3;
	}
	FILE *fp;
	fp = fopen(filename, "wb");
	fprintf(fp, "P%d\n", tp);
	fprintf(fp, "# Created by %s\n", USER_NAME);
	fprintf(fp, "%d %d\n", img->x, img->y);
	fprintf(fp, "%d\n", 255);
	int i;
	if (tp == 3) {
		for (i = 0; i < (img->x*img->y); i++) {
			fprintf(fp, "%d %d %d   ", img->d[i].r, img->d[i].g, img->d[i].b);
			if ((i + 1) % img->x == 0) {
				fprintf(fp, "\n");
			}
		}
	}
	else {

		for (i = 0; i < (img->x*img->y); i++) {
			fwrite(&img->d[i].r, sizeof(unsigned char), 1, fp);
			fwrite(&img->d[i].g, sizeof(unsigned char), 1, fp);
			fwrite(&img->d[i].b, sizeof(unsigned char), 1, fp);
		}
	}

	

	fclose(fp);
}
void runCPU(int *ModeAllred, int *ModeAllgreen, int *ModeAllblue) {
	unsigned short row_quot = 0;
	unsigned short row_rema = 0;
	unsigned short col_quot = 0;
	unsigned short col_rema = 0;
	unsigned short row_limit = 0;
	unsigned short col_limit = 0;
	unsigned short cell_row_limit = 0;
	unsigned short cell_col_limit = 0;
	unsigned long sumr = 0;
	unsigned long sumg = 0;
	unsigned long sumb = 0;
	unsigned long allsumr = 0, allsumg = 0, allsumb = 0;
	signed short y_in_cell, y_in_col, x_in_row, x_in_cell;


	int aveRed = 0, aveGreen = 0, aveBlue = 0, allaver = 0, allaveg = 0, allaveb = 0;

	int i;

	row_quot = image->x / c;
	row_rema = image->y % c;
	col_quot = image->x / c;
	col_rema = image->y % c;

	int cells_per_row;
	int cells_per_col;

	if (row_rema)
		cells_per_row = row_quot + 1;
	else
		cells_per_row = row_quot;

	if (col_rema)
		cells_per_col = col_quot + 1;
	else
		cells_per_col = col_quot;


	for (y_in_col = 0; y_in_col < cells_per_col; y_in_col++)
	{
		if (y_in_col == col_quot)
			col_limit = col_rema;
		else
			col_limit = c;
		for (x_in_row = 0; x_in_row < cells_per_row; x_in_row++)
		{
			if (x_in_row == row_quot)
				row_limit = row_rema;
			else
				row_limit = c;

			sumr = 0;
			sumg = 0;
			sumb = 0;
			for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
			{
				for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
				{
					int temp;
					temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
					sumr += image->d[temp].r;
					sumg += image->d[temp].g;
					sumb += image->d[temp].b;
				}
			}

			for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
			{
				for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
				{
					int temp;
					temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
					image->d[temp].r = (int)sumr / (col_limit*row_limit);
					image->d[temp].g = (int)sumg / (col_limit*row_limit);
					image->d[temp].b = (int)sumb / (col_limit*row_limit);
				}
			}
			allsumr += sumr / (col_limit*row_limit);
			allsumg += sumg / (col_limit*row_limit);
			allsumb += sumb / (col_limit*row_limit);
		}
	}

	allaver = allsumr / (cells_per_row*cells_per_col);
	allaveg = allsumg / (cells_per_row*cells_per_col);
	allaveb = allsumb / (cells_per_row*cells_per_col);

	*ModeAllred = allaver;
	*ModeAllgreen = allaveg;
	*ModeAllblue = allaveb;

}

void runOPENMP(int *ModeAllred, int *ModeAllgreen, int *ModeAllblue) {
	unsigned short row_quot = 0;
	unsigned short row_rema = 0;
	unsigned short col_quot = 0;
	unsigned short col_rema = 0;
	unsigned short row_limit = 0;
	unsigned short col_limit = 0;
	unsigned short cell_row_limit = 0;
	unsigned short cell_col_limit = 0;
	unsigned long sumr = 0;
	unsigned long sumg = 0;
	unsigned long sumb = 0;
	unsigned long allsumr = 0, allsumg = 0, allsumb = 0;
	signed short y_in_cell, y_in_col, x_in_row, x_in_cell;


	int aveRed = 0, aveGreen = 0, aveBlue = 0, allaver = 0, allaveg = 0, allaveb = 0;

	int i;

	row_quot = image->x / c;
	row_rema = image->y % c;
	col_quot = image->x / c;
	col_rema = image->y % c;

	int cells_per_row;
	int cells_per_col;

	if (row_rema)
		cells_per_row = row_quot + 1;
	else
		cells_per_row = row_quot;

	if (col_rema)
		cells_per_col = col_quot + 1;
	else
		cells_per_col = col_quot;


#pragma omp parallel private(y_in_cell, y_in_col, x_in_row, x_in_cell, sumr, sumg, sumb) reduction(+: allsumr, allsumg, allsumb)
	{
#pragma omp for
		for (y_in_col = 0; y_in_col < cells_per_col; y_in_col++)
		{
			if (y_in_col == col_quot)
				col_limit = col_rema;
			else
				col_limit = c;

			for (x_in_row = 0; x_in_row < cells_per_row; x_in_row++)
			{
				if (x_in_row == row_quot)
					row_limit = row_rema;
				else
					row_limit = c;

				sumr = 0;
				sumg = 0;
				sumb = 0;

				for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
				{
					//#pragma omp parallel for reduction(+: sumr,sumg,sumb)
					for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
					{
						int temp;
						temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
						//#pragma omp critical
												//{
#pragma omp atomic 
						sumr += image->d[temp].r;
#pragma omp atomic 
						sumg += image->d[temp].g;
#pragma omp atomic 
						sumb += image->d[temp].b;
						//}
					}
				}

				for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
				{
					//#pragma omp parallel for
					for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
					{
						int temp;
						temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
						//#pragma omp critical
						//						{

						image->d[temp].r = (int)sumr / (col_limit*row_limit);

						image->d[temp].g = (int)sumg / (col_limit*row_limit);

						image->d[temp].b = (int)sumb / (col_limit*row_limit);
						//}
					}
				}
				//#pragma omp atomic 
				allsumr += sumr / (col_limit*row_limit);
				allsumg += sumg / (col_limit*row_limit);
				allsumb += sumb / (col_limit*row_limit);
			}
		}
	}
	//calculate the whole image average value
	allaver = allsumr / (cells_per_row*cells_per_col);
	allaveg = allsumg / (cells_per_row*cells_per_col);
	allaveb = allsumb / (cells_per_row*cells_per_col);

	*ModeAllred = allaver;
	*ModeAllgreen = allaveg;
	*ModeAllblue = allaveb;
}

__device__ unsigned long long int d_sumr, d_sumg, d_sumb;

__device__ void cal_cell_sum(EveryPixel *input,EveryPixel *output, int i, int j,unsigned int *dc, unsigned int *width) {

	int temp;
	unsigned long sumr = 0;
	unsigned long sumg = 0;
	unsigned long sumb = 0;

	unsigned long allsumr = 0;

	unsigned long aver = 0;
	unsigned long aveg = 0;
	unsigned long aveb = 0;

	for (int k = 0; k < *dc; k++) {
		for (int l = 0; l < *dc; l++) {
			temp = j * *dc * *width + k * *width + (i* *dc + l);
			sumr += input[temp].r;
			sumg += input[temp].g;
			sumb += input[temp].b;


		}
	}
	aver = sumr / (*dc* *dc);
	aveg = sumg / (*dc* *dc);
	aveb = sumb / (*dc* *dc);
	//d_sumr[j*(*width / *dc) +i] = aver;
	//d_sumg[j*(*width / *dc) + i] = aveg;
	//d_sumb[j*(*width / *dc) + i] = aveb;


	for (int k = 0; k < *dc; k++) {
		for (int l = 0; l < *dc; l++) {
			temp = j * *dc * *width + k * *width + (i* *dc + l);
			output[temp].r = aver;
			output[temp].g = aveg;
			output[temp].b = aveb;
		}
	}
	__syncthreads();

	atomicAdd(&d_sumr, sumr);
	atomicAdd(&d_sumg, sumg);
	atomicAdd(&d_sumb, sumb);

}

__global__ void runCuda(EveryPixel *input, EveryPixel *output, unsigned int *dc, unsigned int *width) {
	

	cal_cell_sum( input, output, blockIdx.x , threadIdx.x, dc, width);
	

}

void runGPU(int *ModeAllred, int *ModeAllgreen, int *ModeAllblue) {
	unsigned short row_quot = 0;
	unsigned short row_rema = 0;
	unsigned short col_quot = 0;
	unsigned short col_rema = 0;
	unsigned short row_limit = 0;
	unsigned short col_limit = 0;
	unsigned short cell_row_limit = 0;
	unsigned short cell_col_limit = 0;
	unsigned long *sumr;
	unsigned long *sumg;
	unsigned long *sumb;
	unsigned long allsumr = 0, allsumg = 0, allsumb = 0;
	signed short y_in_cell, y_in_col, x_in_row, x_in_cell;
	signed short *dy_in_cell, *dy_in_col, *dx_in_row, *dx_in_cell;
	//unsigned long *d_sumr;
	//unsigned long *d_sumg;
	//unsigned long *d_sumb;

	unsigned long long int sumr2 = 0;
	unsigned long long int sumg2 = 0;
	unsigned long long int sumb2 = 0;

	EveryPixel *d_image, *d_image_out;
	unsigned int *dc, *length, *width;




	int aveRed = 0, aveGreen = 0, aveBlue = 0, allaver = 0, allaveg = 0, allaveb = 0;

	int i;

	row_quot = image->x / c;
	row_rema = image->y % c;
	col_quot = image->x / c;
	col_rema = image->y % c;

	int cells_per_row;
	int cells_per_col;

	if (row_rema)
		cells_per_row = row_quot + 1;
	else
		cells_per_row = row_quot;

	if (col_rema)
		cells_per_col = col_quot + 1;
	else
		cells_per_col = col_quot;

	float time;
	hipEvent_t dstart, stop;
	hipEventCreate(&dstart);
	hipEventCreate(&stop);


	sumr = (unsigned long *)malloc(sizeof(unsigned long *)*cells_per_col*cells_per_row);
	sumg = (unsigned long *)malloc(sizeof(unsigned long *)*cells_per_col*cells_per_row);
	sumb = (unsigned long *)malloc(sizeof(unsigned long *)*cells_per_col*cells_per_row);

	hipMalloc((void **)&dc, sizeof(unsigned int));
	hipMalloc((void **)&length, sizeof(unsigned int));
	hipMalloc((void **)&width, sizeof(unsigned int));

	hipMalloc((void **)&d_image, image->x*image->y * sizeof(EveryPixel));
	hipMalloc((void **)&d_image_out, image->x*image->y * sizeof(EveryPixel));
	hipMalloc((void **)&dy_in_cell, sizeof(signed short));

	hipMalloc((void **)&dx_in_cell, sizeof(signed short));

	//hipMalloc((void **)&d_sumr, sizeof(unsigned long*)*cells_per_col*cells_per_row);
	//hipMalloc((void **)&d_sumg, sizeof(unsigned long*)*cells_per_col*cells_per_row);
	//hipMalloc((void **)&d_sumb, sizeof(unsigned long*)*cells_per_col*cells_per_row);





	checkCUDAError("Memory allocation");

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_sumr), &sumr2, sizeof(unsigned long long int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_sumg), &sumg2, sizeof(unsigned long long int)));
	hipMemcpyToSymbol(HIP_SYMBOL(d_sumb), &sumb2, sizeof(unsigned long long int));

	hipMemcpy(dc, &c, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(length, &image->y, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width, &image->x, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dy_in_cell, &y_in_cell, 1, hipMemcpyHostToDevice);
	hipMemcpy(dx_in_cell, &x_in_cell, 1, hipMemcpyHostToDevice);
	hipMemcpy(d_image, image->d, image->x*image->y * sizeof(EveryPixel), hipMemcpyHostToDevice);



	checkCUDAError("Input transfer to device");



	dim3 blocksPerGrid(cells_per_col, 1, 1);
	dim3 threadsPerBlock(cells_per_row, 1, 1);

	hipEventRecord(dstart, 0);

	runCuda << <blocksPerGrid, threadsPerBlock >> > (d_image, d_image_out, dc, width);

	hipEventRecord(stop, 0);
	hipEventSynchronize(dstart);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, dstart, stop);
	printf("CUDA mode execution time took:\t%f(ms)\n", time);
	hipEventDestroy(dstart);
	hipEventDestroy(stop);

	hipDeviceSynchronize();
	checkCUDAError("Kernel execution");

	hipMemcpy(image->d, d_image_out, image->x*image->y * sizeof(EveryPixel), hipMemcpyDeviceToHost);
	//hipMemcpy(sumr, d_sumr, sizeof(unsigned long *)*cells_per_col*cells_per_row, hipMemcpyDeviceToHost);
	//hipMemcpy(sumg, d_sumg, sizeof(unsigned long *)*cells_per_col*cells_per_row, hipMemcpyDeviceToHost);
	//hipMemcpy(sumb, d_sumb, sizeof(unsigned long *)*cells_per_col*cells_per_row, hipMemcpyDeviceToHost);

	hipMemcpyFromSymbol(&sumr2, HIP_SYMBOL(d_sumr), sizeof(unsigned long long int));
	hipMemcpyFromSymbol(&sumg2, HIP_SYMBOL(d_sumg), sizeof(unsigned long long int));
	hipMemcpyFromSymbol(&sumb2, HIP_SYMBOL(d_sumb), sizeof(unsigned long long int));

	long long int showaver = 0, showaveg = 0, showaveb = 0;
	//for (int i = 0; i < cells_per_col*cells_per_row; i++) {
	//	showaver += sumr[i];
	//	showaveg += sumg[i];
	//	showaveb += sumb[i];
	//}
	showaver = sumr2;
	showaveg = sumg2;
	showaveb = sumb2;

	allaver = showaver / (image->x*image->y);
	allaveg = showaveg / (image->x*image->y);
	allaveb = showaveb / (image->x*image->y);
	*ModeAllred = allaver;
	*ModeAllgreen = allaveg;
	*ModeAllblue = allaveb;

	hipFree(d_image);
	hipFree(d_image_out);
	//hipFree(d_sumr);
	//hipFree(d_sumg);
	//hipFree(d_sumb);
	hipFree(dc);
	hipFree(length);
	hipFree(width);
	hipFree(dy_in_cell);
	hipFree(dx_in_cell);
	checkCUDAError("Free memory");
}
int main(int argc, char *argv[]) {
	int ModeAllRed = 0;
	int ModeAllGreen = 0;
	int ModeAllBlue = 0;

	if (process_command_line(argc, argv) == FAILURE)
		return 1;
	//if c is the power of 2.
	if (!((c > 0) && ((c & (c - 1)) == 0)))
	{
		printf("Entry power of 2 number");
		return 1;
	}

	if (!strcmp(modename, "CPU")) {
		execution_mode = CPU;
	}
	else
	{
		if (!strcmp(modename, "OPENMP")) {
			execution_mode = OPENMP;
		}
		else {
			if (!strcmp(modename, "CUDA")) {
				execution_mode = CUDA;
			}
			else {
				if (!strcmp(modename, "ALL")) {
					execution_mode = ALL;
				}
				else {
					printf("Enter a right mode");
					exit(1);
				}
			}
		}
	}

	//TODO: read input image file (either binary or plain text PPM) 

	image = (WholeImage *)malloc(sizeof(WholeImage));
	readFile();

	//TODO: execute the mosaic filter based on the mode
	switch (execution_mode) {
	case (CPU): {

		//TODO: starting timing here
		clock_t start, end;
		start = clock();
		double total;
		
		unsigned short row_quot = 0;
		unsigned short row_rema = 0;
		unsigned short col_quot = 0;
		unsigned short col_rema = 0;
		unsigned short row_limit = 0;
		unsigned short col_limit = 0;
		unsigned short cell_row_limit = 0;
		unsigned short cell_col_limit = 0;
		unsigned long sumr = 0;
		unsigned long sumg = 0;
		unsigned long sumb = 0;
		unsigned long allsumr = 0, allsumg = 0, allsumb = 0;
		signed short y_in_cell, y_in_col, x_in_row, x_in_cell;

	
		int aveRed = 0, aveGreen = 0, aveBlue = 0, allaver = 0, allaveg = 0, allaveb = 0;

		int i;

		row_quot = image->x / c;
		row_rema = image->y % c;
		col_quot = image->x / c;
		col_rema = image->y % c;

		int cells_per_row;
		int cells_per_col;

		if (row_rema)
			cells_per_row = row_quot + 1;
		else
			cells_per_row = row_quot;

		if (col_rema)
			cells_per_col = col_quot + 1;
		else
			cells_per_col = col_quot;
		
		
		for ( y_in_col = 0; y_in_col < cells_per_col; y_in_col++)
		{
			if (y_in_col == col_quot)
				col_limit = col_rema;
			else
				col_limit = c;
			for (x_in_row = 0; x_in_row < cells_per_row; x_in_row++)
			{
				if (x_in_row == row_quot)
					row_limit = row_rema;
				else
					row_limit = c;

				sumr = 0;
				sumg = 0;
				sumb = 0;
				for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
				{
					for ( x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
					{
						int temp;
						temp = y_in_col * (c*image->x) + y_in_cell*image->x + (x_in_row*c + x_in_cell);
						sumr += image->d[temp].r;
						sumg += image->d[temp].g;
						sumb += image->d[temp].b;
					}
				}

				for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
				{
					for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
					{
						int temp;
						temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
						image->d[temp].r = (int)sumr / (col_limit*row_limit);
						image->d[temp].g = (int)sumg / (col_limit*row_limit);
						image->d[temp].b = (int)sumb / (col_limit*row_limit);
					}
				}
				allsumr += sumr / (col_limit*row_limit);
				allsumg += sumg / (col_limit*row_limit);
				allsumb += sumb / (col_limit*row_limit);
			}
		}

		allaver = allsumr / (cells_per_row*cells_per_col);
		allaveg = allsumg / (cells_per_row*cells_per_col);
		allaveb = allsumb / (cells_per_row*cells_per_col);
		printf("CPU Average image colour red = %d, green = %d, blue = %d \n", allaver, allaveg, allaveb);

		//TODO: end timing here
		end = clock();
		total = (double)(end - start) / CLOCKS_PER_SEC;
		int a = total;
		double ms = (double)(total - a) * 1000;
		printf("CPU mode execution time took %d s and %f ms\n", a, ms);
		break;
	}
	case (OPENMP): {
		//TODO: starting timing here
		double start = omp_get_wtime();
		double total;

		unsigned short row_quot = 0;
		unsigned short row_rema = 0;
		unsigned short col_quot = 0;
		unsigned short col_rema = 0;
		unsigned short row_limit = 0;
		unsigned short col_limit = 0;
		unsigned short cell_row_limit = 0;
		unsigned short cell_col_limit = 0;
		unsigned long sumr = 0;
		unsigned long sumg = 0;
		unsigned long sumb = 0;
		unsigned long allsumr = 0, allsumg = 0, allsumb = 0;
		signed short y_in_cell, y_in_col, x_in_row, x_in_cell;


		int aveRed = 0, aveGreen = 0, aveBlue = 0, allaver = 0, allaveg = 0, allaveb = 0;

		int i;

		row_quot = image->x / c;
		row_rema = image->y % c;
		col_quot = image->x / c;
		col_rema = image->y % c;

		int cells_per_row;
		int cells_per_col;

		if (row_rema)
		cells_per_row = row_quot + 1;
		else
		cells_per_row = row_quot;

		if (col_rema)
		cells_per_col = col_quot + 1;
		else
		cells_per_col = col_quot;


#pragma omp parallel private(y_in_cell, y_in_col, x_in_row, x_in_cell, sumr, sumg, sumb) reduction(+: allsumr, allsumg, allsumb)
		{
#pragma omp for
			for (y_in_col = 0; y_in_col < cells_per_col; y_in_col++)
			{
				if (y_in_col == col_quot)
					col_limit = col_rema;
				else
					col_limit = c;

				for (x_in_row = 0; x_in_row < cells_per_row; x_in_row++)
				{
					if (x_in_row == row_quot)
						row_limit = row_rema;
					else
						row_limit = c;

					sumr = 0;
					sumg = 0;
					sumb = 0;

					for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
					{
						//#pragma omp parallel for reduction(+: sumr,sumg,sumb)
						for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
						{
							int temp;
							temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
							//#pragma omp critical
													//{
#pragma omp atomic 
							sumr += image->d[temp].r;
#pragma omp atomic 
							sumg += image->d[temp].g;
#pragma omp atomic 
							sumb += image->d[temp].b;
							//}
						}
					}

					for (y_in_cell = 0; y_in_cell < col_limit; y_in_cell++)
					{
						//#pragma omp parallel for
						for (x_in_cell = 0; x_in_cell < row_limit; x_in_cell++)
						{
							int temp;
							temp = y_in_col * (c*image->x) + y_in_cell * image->x + (x_in_row*c + x_in_cell);
							//#pragma omp critical
							//						{

							image->d[temp].r = (int)sumr / (col_limit*row_limit);

							image->d[temp].g = (int)sumg / (col_limit*row_limit);

							image->d[temp].b = (int)sumb / (col_limit*row_limit);
							//}
						}
					}
//#pragma omp atomic 
					allsumr += sumr / (col_limit*row_limit);
					allsumg += sumg / (col_limit*row_limit);
					allsumb += sumb / (col_limit*row_limit);
				}
			}
		}
		//calculate the whole image average value
		allaver = allsumr / (cells_per_row*cells_per_col);
		allaveg = allsumg / (cells_per_row*cells_per_col);
		allaveb = allsumb / (cells_per_row*cells_per_col);


//		// Output the average colour value for the image
		printf("OPENMP Average image colour red = %d, green = %d, blue = %d \n", allaver, allaveg, allaveb);
//
//		//TODO: end timing here
		double end = omp_get_wtime();
		total = end - start;
		int a = total;
		double ms = (double)(total - a) * 1000;
		printf("OPENMP mode execution time took %d s and %f ms\n", a, ms);
		break;
	}
	case (CUDA): {

		runGPU(&ModeAllRed, &ModeAllGreen, &ModeAllBlue);
		printf("CUDA Average image colour red = %d, green = %d, blue = %d \n", ModeAllRed, ModeAllGreen, ModeAllBlue);
		break;
	}
	case (ALL): {

		clock_t start, end;
		start = clock();
		double total;

		runCPU(&ModeAllRed, &ModeAllGreen, &ModeAllBlue);
		printf("CPU Average image colour red = %d, green = %d, blue = %d \n", ModeAllRed, ModeAllGreen, ModeAllBlue);

		end = clock();
		total = (double)(end - start) / CLOCKS_PER_SEC;
		int a = total;
		double ms = (double)(total - a) * 1000;
		printf("CPU mode execution time took %d s and %f ms\n", a, ms);

		readFile();
		double start2 = omp_get_wtime();
		double total2;

		runOPENMP(&ModeAllRed, &ModeAllGreen, &ModeAllBlue);
		printf("OPENMP Average image colour red = %d, green = %d, blue = %d \n", ModeAllRed, ModeAllGreen, ModeAllBlue);

		double end2 = omp_get_wtime();
		total2 = end2 - start2;
		int a2 = total2;
		double ms2 = (double)(total2 - a2) * 1000;
		printf("OPENMP mode execution time took %d s and %f ms\n", a2, ms2);

		readFile();


		
		runGPU(&ModeAllRed, &ModeAllGreen, &ModeAllBlue);
		printf("CUDA Average image colour red = %d, green = %d, blue = %d \n", ModeAllRed, ModeAllGreen, ModeAllBlue);
		/*hipFree(d_image);
		hipFree(d_image_out);
		hipFree(d_sumr);
		hipFree(d_sumg);
		hipFree(d_sumb);
		hipFree(dc);
		hipFree(length);
		hipFree(width);
		hipFree(dy_in_cell);
		hipFree(dx_in_cell);
		checkCUDAError("Free memory");*/

		break;
	}
	}

	writeFile(outname, image, c, option);
	free(image);
	//save the output image file (from last executed mode) 
	return 0;
}

void print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP, CUDA or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");
}

int process_command_line(int argc, char *argv[]) {
	if (argc < 7) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}

	//first argument is always the executable name

	//read in the non optional command line arguments
	c = (unsigned int)atoi(argv[1]);

	//TODO: read in the mode
	modename = argv[2];
	//TODO: read in the input image name
	filename = argv[4];
	//TODO: read in the output image name
	outname = argv[6];
	//TODO: read in any optional part 3 arguments
	if (argc == 7) {
		option = NULL;
	}
	else
		option = argv[7];
	
	return SUCCESS;
}

int readFile() {
	FILE * f = fopen(filename, "rb");
	if (f == NULL)
	{
		printf("cannot open");
		return 1;
	}


	image = (WholeImage *)malloc(sizeof(WholeImage));
	if (!fgets(buffer, sizeof(buffer), f)) {
		printf("error");
		exit(1);
	}
	char type = buffer[1];
	int comments, rgbcolor;
	switch (type)
	{
	case('3'): {
		comments = getc(f);
		while (comments == '#')
		{
			while (getc(f) != '\n');
			comments = getc(f);
		}
		ungetc(comments, f);
		fscanf(f, "%d %d", &image->x, &image->y);
		fscanf(f, "%d", &rgbcolor);
		while (fgetc(f) != '\n');
		image->d = (EveryPixel *)malloc(image->x*image->y * sizeof(EveryPixel));
		for (int i = 0; i < (image->x*image->y); i++) {
			fscanf(f, "%d%d%d", &image->d[i].r, &image->d[i].g, &image->d[i].b);
		}
		fclose(f);
		break;
	}
	case('6'): {
		comments = getc(f);
		while (comments == '#')
		{
			while (getc(f) != '\n');
			comments = getc(f);
		}
		ungetc(comments, f);
		fscanf(f, "%d %d", &image->x, &image->y);
		fscanf(f, "%d", &rgbcolor);
		while (fgetc(f) != '\n');
		image->d = (EveryPixel *)malloc(image->x*image->y * sizeof(EveryPixel));
		fread(image->d, 3 * image->x, image->y, f);
		fclose(f);
		break;
	}
	default:
		break;
	}
	return 0;
}


